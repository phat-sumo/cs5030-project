// Shared GPU implementation
#include <cassert>
#include <stdio.h>
#include <stdint.h>
#include <stdbool.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#include <hip/hip_runtime.h>

#include "../common/cuda_bresenham.cu"

int main() {
	const char input_filename[] = "../common/srtm_14_04_6000x6000_short16.raw";
	const char output_filename[] = "../output/srtm_14_04_shared_gpu_out_6000x6000_uint32.raw";
	// const char input_filename[] = "../common/srtm_14_04_300x300_short16.raw";
	// const char output_filename[] = "../output/srtm_14_04_shared_gpu_out_300x300_uint32.raw";

	// Initialize map data
	const int width = 6000;
	const int height = 6000;
	// const int width = 300;
	// const int height = 300;
	const int num_values = width * height;

	// Open file containing elevation data
	FILE* input_file = fopen(input_filename, "r");

	if (input_file == NULL) {
		printf("could not open file %s\n", input_filename);
		return 1;
	}

	// Read in elevation data
	short* h_values = (short*) malloc(num_values * sizeof(short));
	fread(h_values, sizeof(short), num_values * sizeof(short), input_file);
	fclose(input_file);

	// Set all output elements to 0
	uint32_t* h_output = (uint32_t*) malloc(num_values * sizeof(uint32_t));
	memset(h_output, 0, num_values * sizeof(uint32_t));

	// Allocate cuda device variables
    short* d_values = NULL;
	uint32_t* d_output = NULL;
    checkCuda(hipMalloc((void **)&d_values, num_values * sizeof(short)));
	checkCuda(hipMalloc((void **)&d_output, num_values * sizeof(uint32_t)));
    checkCuda(hipMemcpy(d_values, h_values, num_values * sizeof(short), hipMemcpyHostToDevice));

	// Begin execution timing
	struct timespec ts_start;
	clock_gettime(CLOCK_MONOTONIC, &ts_start);

	// Initialize the grid and block sizes from the map width and height
    const dim3 grid_size(width / 16, height / 16, 1);
    const dim3 block_size(16, 16, 1);
    cuda_bresenham<<<grid_size, block_size>>>(width, height, d_values, d_output);

	// Synchronize and transfer the results from the device back to the host
    checkCuda(hipDeviceSynchronize());
    checkCuda(hipMemcpy(h_output, d_output, num_values * sizeof(uint32_t), hipMemcpyDeviceToHost));

	// End execution timing
	struct timespec ts_end;
	clock_gettime(CLOCK_MONOTONIC, &ts_end);
	printf("Total elapsed time: %ld\n", (ts_end.tv_sec - ts_start.tv_sec) * 1000000);

	// Write output data to file
	FILE* output_file = fopen(output_filename, "w");
	fwrite(h_output, sizeof(unsigned char), num_values * sizeof(uint32_t), output_file);
	fclose(output_file);

	// Clean up allocated variables
	free(h_values);
	free(h_output);
	hipFree(d_values);
	hipFree(d_output);

	return 0;
}
