// Distributed GPU implementation
#include <cassert>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <mpi.h>
#include <hip/hip_runtime.h>

#include "../common/cuda_bresenham.cu"

#define CLOCK_MONOTONIC 1

int main(int argc, char** argv) {
  // Process the 6000x6000 data by default
  char input_filename[] = "../common/srtm_14_04_6000x6000_short16.raw";
	char output_filename[] = "../common/srtm_14_04_out_6000x6000_uint32.raw";
	// char input_filename[] = "../common/srtm_14_04_300x300_short16.raw";
	// char output_filename[] = "../output/srtm_14_04_distributed_gpu_out_300x300_uint32.raw";

  const int width = 6000;
  const int height = 6000;
  // const int width = 300;
  // const int height = 300;

  // Initialize MPI
  MPI_Init(&argc, &argv);

  // Initialize MPI size and rank
  int rank, size;
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  MPI_Comm_size(MPI_COMM_WORLD, &size);

  // Associate each MPI rank with a GPU device (assumes single node - multi gpu)
  int num_devices = 0;
  checkCuda(hipGetDeviceCount(&num_devices));
  checkCuda(hipSetDevice(rank % num_devices));
  checkCuda(hipFree(0));

  // Instantiate a map struct
  ElevationMap map;
	map.width = width;
	map.height = height;
  const int map_size = map.width * map.height;
  map.values = (short*) malloc(map_size * sizeof(short));

  // Instantiate a bounds struct
  Bounds bounds_local;
	get_bounds(map, size, rank, &bounds_local);

  // Cuda device variables
  short* d_values = NULL;
	uint32_t* d_output = NULL;

  // Allocate a differently sized output depending on the rank
  uint32_t* h_output = NULL;

  if (rank == 0) {
    // Open file containing elevation data
    FILE* input_file = fopen(input_filename, "r");
    if (input_file == NULL) {
		  printf("could not open file %s\n", input_filename);
		  return 1;
	  }

    // Read in elevation data
    fread(map.values, sizeof(short), map_size * sizeof(short), input_file);
    fclose(input_file);

    // Set all output elements to 0
    h_output = (uint32_t*) malloc(map_size * sizeof(uint32_t));
		memset(h_output, 0, map_size * sizeof(uint32_t));

    // Distribute elevation data across MPI processes
		for (int rank = 1; rank < size; rank++) {
			Bounds b;
			get_bounds(map, size, rank, &b);
			MPI_Send(map.values + b.start, b.length, MPI_SHORT, rank, 0, MPI_COMM_WORLD);
		}

    // Allocate CUDA device variables
    checkCuda(hipMalloc((void **)&d_values, bounds_local.slice_size * sizeof(short)));
	  checkCuda(hipMalloc((void **)&d_output, map_size * sizeof(uint32_t)));
    checkCuda(hipMemcpy(d_values, map.values, bounds_local.slice_size * sizeof(short), hipMemcpyHostToDevice));

    // Begin execution timing
    struct timespec ts_start;
		clock_gettime(CLOCK_MONOTONIC, &ts_start);

    // Compute partial viewshed from elevation map
    const dim3 grid_size(width / 16, height / 16, 1);
  	const dim3 block_size(16, 16, 1);
	  cuda_bresenham<<<grid_size, block_size>>>(width, height, d_values, d_output);

    // Synchronize and transfer the results from the device back to the host
    checkCuda(hipDeviceSynchronize());
    checkCuda(hipMemcpy(h_output, d_output, bounds_local.slice_size * sizeof(uint32_t), hipMemcpyDeviceToHost));

    // Wait to recieve all partial outputs from other processes
		for (int rank = 1; rank < size; rank++) {
			Bounds b;
			get_bounds(map, size, rank, &b);
			MPI_Recv(h_output + b.offset, b.slice_size, MPI_UINT32_T, rank, 1, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
		}

    // End execution timing
    struct timespec ts_end;
		clock_gettime(CLOCK_MONOTONIC, &ts_end);
		printf("Total elapsed time: %ld\n", (ts_end.tv_sec - ts_start.tv_sec));

		// Write output data to file
		FILE* output_file = fopen(output_filename, "w");
		fwrite(h_output, sizeof(unsigned char), map_size * sizeof(uint32_t), output_file);
		fclose(output_file);
  } else {
    // Allocate partial output
    h_output = (uint32_t *) malloc(sizeof(uint32_t) * bounds_local.slice_size);

    // Recieve partial map values from rank 0
    MPI_Recv(map.values + bounds_local.start, bounds_local.length, MPI_SHORT, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
		printf("%d: received message\n", rank);

    // Allocate CUDA device variables
    checkCuda(hipMalloc((void **)&d_values, bounds_local.slice_size * sizeof(short)));
	  checkCuda(hipMalloc((void **)&d_output, map_size * sizeof(uint32_t)));
    checkCuda(hipMemcpy(d_values, map.values, bounds_local.slice_size * sizeof(short), hipMemcpyHostToDevice));

    // Compute partial viewshed from elevation map
    const dim3 grid_size(width / 16, height / 16, 1);
  	const dim3 block_size(16, 16, 1);
	  cuda_bresenham<<<grid_size, block_size>>>(width, height, d_values, d_output);

    // Synchronize and transfer the results from the device back to the host
    checkCuda(hipDeviceSynchronize());
    checkCuda(hipMemcpy(h_output, d_output, bounds_local.slice_size * sizeof(uint32_t), hipMemcpyDeviceToHost));

    // Send computed results back to rank 0
		MPI_Send(h_output, bounds_local.slice_size, MPI_UINT32_T, 0, 1, MPI_COMM_WORLD);
  }

  // Wait for all processes before ending the program
  MPI_Barrier(MPI_COMM_WORLD);

  // Clean up allocated variables
  hipFree(d_values);
  hipFree(d_output);
  free(map.values);
  free(h_output);

  // Clean up MPI
  MPI_Finalize();

  return 0;
}
